// Experience replay buffer on GPU

#include <hip/hip_runtime.h>

struct Experience {
    float* states;
    int* actions;
    float* rewards;
    float* next_states;
    bool* dones;
    float* log_probs;
    float* values;
};

class ReplayBuffer {
public:
    ReplayBuffer(int capacity, int state_dim, int num_envs);
    ~ReplayBuffer();

    void add(float* states, int* actions, float* rewards,
             float* next_states, bool* dones, float* log_probs, float* values);

    void sample(int batch_size, Experience& batch);
    void clear();

    int size();
    bool isFull();

private:
    float* d_states;
    int* d_actions;
    float* d_rewards;
    float* d_next_states;
    bool* d_dones;
    float* d_log_probs;
    float* d_values;

    int capacity;
    int current_size;
    int write_idx;
    int state_dim;
};
