// Custom CUDA optimizer (Adam)

#include <hip/hip_runtime.h>

struct OptimizerState {
    float* m;  // First moment estimate
    float* v;  // Second moment estimate
    int t;     // Timestep
};

// Adam optimizer kernel
__global__ void adamStep(float* params, float* grads,
                        float* m, float* v, int t,
                        float lr, float beta1, float beta2, float epsilon,
                        int num_params) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_params) return;

    // Update biased first moment estimate
    m[idx] = beta1 * m[idx] + (1.0f - beta1) * grads[idx];

    // Update biased second moment estimate
    v[idx] = beta2 * v[idx] + (1.0f - beta2) * grads[idx] * grads[idx];

    // Compute bias-corrected moment estimates
    float m_hat = m[idx] / (1.0f - powf(beta1, t));
    float v_hat = v[idx] / (1.0f - powf(beta2, t));

    // Update parameters
    params[idx] -= lr * m_hat / (sqrtf(v_hat) + epsilon);
}

class AdamOptimizer {
public:
    AdamOptimizer(int num_params, float lr = 3e-4f);
    void step(float* params, float* grads);
    void reset();

private:
    OptimizerState state;
    float learning_rate;
};
