// Neural Network Implementation in Pure CUDA
// Policy and Value networks

#include <hip/hip_runtime.h>

// Network architecture
struct NetworkConfig {
    int input_size;
    int hidden_sizes[4];
    int num_hidden_layers;
    int policy_output_size;
    int value_output_size;
};

// Forward pass kernels
__global__ void linearForward(float* input, float* weights, float* bias,
                             float* output, int in_size, int out_size, int batch_size);

__global__ void reluActivation(float* x, int size);

__global__ void softmaxForward(float* logits, float* probs, int batch_size, int num_actions);

// Backward pass kernels
__global__ void linearBackward(float* grad_output, float* grad_input,
                              float* grad_weights, float* grad_bias,
                              float* input, float* weights,
                              int in_size, int out_size, int batch_size);

__global__ void reluBackward(float* grad_output, float* x, float* grad_input, int size);

// Policy network
class PolicyNetwork {
public:
    PolicyNetwork(NetworkConfig config);
    void forward(float* states, float* action_probs, int batch_size);
    void backward(float* grad_loss, int batch_size);

private:
    float* d_weights[5];
    float* d_biases[5];
    NetworkConfig config;
};

// Value network
class ValueNetwork {
public:
    ValueNetwork(NetworkConfig config);
    void forward(float* states, float* values, int batch_size);
    void backward(float* grad_loss, int batch_size);

private:
    float* d_weights[5];
    float* d_biases[5];
    NetworkConfig config;
};
